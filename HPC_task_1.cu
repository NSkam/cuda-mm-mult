#include "hip/hip_runtime.h"


#include "hipblas.h"
#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <thread>
#include <chrono>
#include <hiprand/hiprand.h>
#pragma comment(lib,"cublas.lib")

using namespace std;

int lines, columns;

void init() {
	bool endc = false;
	bool endl = false;
	while (!endl || !endc)
	{
		printf("Give me the size of lines\n");

		scanf("%d", &lines);
		endl = true;

		if (lines <= 0) {
			printf("You must use a number of lines that is greater than 0!\n");
			endl = false;
		}
		if (endl == true) {
			printf("Give me the size of columns\n");
			scanf("%d", &columns);
			endc = true;
			if (columns <= 0) {
				printf("You must use a number of columns that is greater than 0!\n");
				endc = false;
				columns = 0;
			}
		}
	}
}


int gpu_blas_mmul(const float* A, float* C, const int row, const int col) {
	float time;
	hipEvent_t start, stop;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	//CuBLAS initialization
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS initialization failed\n");
		return EXIT_FAILURE;
	}
		int lda = row, ldb=row, ldc = col;
		const float alf = 1;
		const float bet = 0;
	    const float* alpha = &alf;
	    const float* beta = &bet;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

     //The actual calculation
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, col, col, row, alpha, A, lda, A, ldb, beta, C, ldc);
	// for (int i = 0; i < (columns) * (columns); i++) printf("C[%d] = %.4f\n", i, C[i]);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time for calculations:  %3.6f ms \n", time);

	// Destroy the handle
	 hipblasDestroy(handle);

}

//Fill the matrix with random numbers
 void GPU_fill_rand(float* A, int nr_rows_A, int nr_cols_A) {
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
     hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

 //Print matrix
 void print_matrix(const float* A, int nr_rows_A, int nr_cols_A) { 
	 for (int i = 0; i < nr_rows_A; ++i) {
		 for (int j = 0; j < nr_cols_A; ++j) {
			 printf("C(%d,%d)=%.2f\n", i+1,j+1, A[j * nr_rows_A + i]);
		 }
	 }
 }


int main() {

	hipError_t cudaStat;

	//time variables
	float time;
	hipEvent_t start, stop;

	init();
	float* A = (float*)malloc(lines * columns * sizeof(float));
	float* C = (float*)malloc(columns * columns * sizeof(float));
	//For Matlab Testing
/*	for (int i = 0; i < lines; i++)
		for (int j = 0; j < columns; j++)
			if (j < columns)
				printf("A(%d,%d) = %.4f;\n", i + 1, j + 1, A[i * columns + j]);*/
	float* A_D;
	float* C_D;

	//Allocate memory in device for A
	cudaStat = hipMalloc(&A_D, lines * columns * sizeof(float));
	if (cudaStat != hipSuccess) {
		printf("device memory allocation failed");
		return EXIT_FAILURE;
	}
	//Allocate memory in device for C
	cudaStat = hipMalloc(&C_D, columns * columns * sizeof(float));
	if (cudaStat != hipSuccess) {
		printf("device memory allocation failed");
		return EXIT_FAILURE;
	}  

	GPU_fill_rand(A_D, lines, columns);

	hipMemcpy(A, A_D, lines * columns * sizeof(float), hipMemcpyDeviceToHost);
	//print_matrix(A, lines, columns);

	//Call multiply function
	gpu_blas_mmul(A_D, C_D, lines, columns);
	
	//print the time
	//printf("Time for calculations:  %3.1f ms \n", time);

	// Copy (and print) the result on host memory
	hipMemcpy(C, C_D, (columns*columns * sizeof(float)), hipMemcpyDeviceToHost);
	//print_matrix(C, columns, columns);
	hipFree(A_D);
	hipFree(C_D);
	free(A);
	free(C);
	return 0;
}